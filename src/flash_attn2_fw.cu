#include "hip/hip_runtime.h"
#include <math.h>

#include <cub/block/block_load.cuh>
#include <hipcub/hipcub.hpp>
#include <cmath>
#include "includes/block_reduce.h"
#include "includes/kernels.h"
#include <sys/time.h>


#include <hip/hip_cooperative_groups.h>
#define BASE_THREAD_NUM 32
#define TILE_SIZE 2024
#define MBY4D 16

namespace cg = cooperative_groups;
const float EPSILON = 1e-8f;

namespace lightseq {
namespace cuda {

template <typename T> //, int block_dim, int ele_per_thread>
__global__ void flash_attn_fw(T *q, T *k, T *v, T *out, T *l, T *m, int batch, int N, int d, bool causal_mask=false) {
    
    int batch_idx = blockIdx.x;
    int tidx = threadIdx.x;
    int tidx_y = threadIdx.y;

    //printf("Batch Idx : %d\n", batch_idx);
    q += batch_idx * N * d;
    k +=  batch_idx * N * d;
    v += batch_idx * N * d;
    out += batch_idx * N * d;
    l += batch_idx * N;
    m += batch_idx * N;
    
    float tau = sqrt(1.0/d);    
    int B_c = MBY4D; //BASE_THREAD_NUM; //on_chip_memory_size / (4 * d);  // Using 4 bytes per float
    int B_r = min(B_c, d); //min(on_chip_memory_size / (4 * d), d);
    int T_r = (N + B_r - 1)/ B_r;
    int T_c = (N +B_c -1)/ B_c;

    assert(TILE_SIZE > MBY4D * d);
    //assert(d < TILE_SIZE/BASE_THREAD_NUM);
    __shared__ float sram[TILE_SIZE * 6];
    float* Qi = sram;
    float* Kj = &sram[TILE_SIZE];
    float* Vj = &sram[TILE_SIZE * 2];
    float* Sij  = &sram[TILE_SIZE * 3];
    
    float* tempPRO  = &sram[TILE_SIZE * 4];
    float* Oi  = &sram[TILE_SIZE * 5];

    __shared__ float lm_sram[MBY4D * 3];
    float* li = lm_sram;
    float* mi = &lm_sram[MBY4D];
    float* mi_prev = &lm_sram[MBY4D*2];

    int B_c_blocks = (B_c + BASE_THREAD_NUM - 1)/ BASE_THREAD_NUM;
    int B_r_blocks = (B_r + BASE_THREAD_NUM - 1)/ BASE_THREAD_NUM;
    int d_blocks = (d + BASE_THREAD_NUM - 1)/ BASE_THREAD_NUM;

    
#ifdef TIME
    clock_t start_time, end_time;
#endif    
    int i = blockIdx.y;
    //for(int i = 0; i < T_r; i++){
        // Loading
        for(int read_block=0; read_block < B_r_blocks; read_block++){
            int tidx_ = read_block * BASE_THREAD_NUM + tidx;
            if(tidx_ < B_r){
                li[tidx_] = 0;
                mi[tidx_] = -10000000;
                for(int read_block_y=0; read_block_y < d_blocks; read_block_y++){                   
                    int tidx_y_ = read_block_y * BASE_THREAD_NUM + tidx_y;
                    if(tidx_y_ < d){ 
                        if(i * B_r + tidx_ < N){
                            Qi[tidx_ * d + tidx_y_]  = q[(i * B_r + tidx_) * d + tidx_y_];
                            Oi[tidx_ * d + tidx_y_]  = 0;
                        }
                        else{
                            Qi[tidx_ * d + tidx_y_]  = 0;
                            Oi[tidx_ * d + tidx_y_]  = 0;
                        }
                    }
                }
            }
        }

        __syncthreads();
        
        for(int j = 0; j < T_c; j++){
            // Loading 
#ifdef CAUSAL_BLOCKSPARSE
            if(causal_mask && (j * B_c > (i+1) * B_r -1)){
                continue;
            }
#endif
#ifdef TIME
            if(tidx==0 and tidx_y == 0)
                 start_time  = clock();
#endif

            for(int read_block = 0;read_block < B_c_blocks; read_block++){
                for(int read_block_y = 0; read_block_y < d_blocks; read_block_y++){
                    int tidx_ = read_block * BASE_THREAD_NUM + tidx;
                    int tidx_y_ = read_block_y * BASE_THREAD_NUM + tidx_y;
                    
                    if(tidx_ < B_c && j * B_c + tidx_ < N && tidx_y_ < d){
                        Kj[tidx_ * d + tidx_y_] = k[(j * B_c + tidx_) * d + tidx_y_];
                        Vj[tidx_ * d + tidx_y_] = v[(j * B_c + tidx_) * d + tidx_y_];
                    }
                    else if(tidx_ < B_c && tidx_y == 0){
                        Kj[tidx_ * d + tidx_y_] = 0;
                        Vj[tidx_ * d + tidx_y_] = 0;
                    }
                }
            }
            for(int read_block=0; read_block < B_r_blocks; read_block++){
                int tidx_ = read_block * BASE_THREAD_NUM + tidx;
                if(tidx_ < B_r){
                    for(int read_block_y=0; read_block_y < d_blocks; read_block_y++){                   
                        int tidx_y_ = read_block_y * BASE_THREAD_NUM + tidx_y;
                        if(tidx_y_ < d){ 
                            tempPRO[tidx_ * d + tidx_y_] = 0; 
                        }
                    }
                }
            }
            for(int read_block=0; read_block < B_r_blocks; read_block++){
                for(int read_block_y=0; read_block_y < B_c_blocks; read_block_y++){
                    int tidx_ = read_block * BASE_THREAD_NUM + tidx;
                    int tidx_y_ = read_block_y * BASE_THREAD_NUM + tidx_y;
                    if(tidx_ <B_r &&  tidx_y_ < B_c ){
                        Sij[tidx_ * B_c + tidx_y_] = 0;
                    }   

                }
            }
            

            __syncthreads();
#ifdef TIME
            if(tidx==0 and tidx_y == 0){
                end_time  = clock();
                printf("%f: init\n", 1000000.0*(end_time - start_time));
                start_time = clock();
            }
#endif
            
            for(int read_block=0; read_block < B_r_blocks; read_block++){
                for(int read_block_y=0; read_block_y < B_c_blocks; read_block_y++){
                    int tidx_ = read_block * BASE_THREAD_NUM + tidx;
                    int tidx_y_ = read_block_y * BASE_THREAD_NUM + tidx_y;
                    if(tidx_ <B_r && (i * B_r + tidx_ < N) && tidx_y_ < B_c && (j * B_c + tidx_y_ < N)){
                        if(!causal_mask || j * B_c + tidx_y_ <= i * B_r + tidx_){
                            float S_acc = 0;
                            for(int y = 0; y < d; y++)
                                S_acc += (tau * Qi[tidx_ * d + y] * Kj[tidx_y_ * d + y]);
                            Sij[tidx_ * B_c + tidx_y_]  = S_acc; 
                        }
                        else
                            Sij[tidx_ * B_c + tidx_y_]  = -10000000;
                    }
                }
            }
            __syncthreads();

#ifdef TIME
            if(tidx==0 and tidx_y == 0){
                end_time  = clock();
                printf("%f: S\n", 1000000.0*(end_time - start_time));
                start_time = clock();
            }
#endif

            for(int read_block=0; read_block < B_r_blocks; read_block++){
                int tidx_ = read_block * BASE_THREAD_NUM + tidx;
                if(tidx_ < B_r && (i * B_r + tidx_ < N)  && tidx_y == 0){
                    mi_prev[tidx_] = mi[tidx_];
                }
                for(int y = 0; y < B_c; y++){
                    if(tidx_ < B_r && (i * B_r + tidx_ < N) && tidx_y == 0 && (j * B_c + y < N)){
                        mi[tidx_] = max(mi[tidx_], Sij[tidx_ * B_c + y]);
                    }  
                }
                if(tidx_ < B_r && (i * B_r + tidx_ < N)  && tidx_y == 0){
                    li[tidx_] = exp(mi_prev[tidx_] - mi[tidx_]) * li[tidx_];
                }
            }
            __syncthreads();

#ifdef TIME
            if(tidx==0 and tidx_y == 0){
                end_time  = clock();
                printf("%f: Mij\n", 1000000.0*(end_time - start_time));
                start_time = clock();
            }
#endif
            for(int read_block=0; read_block < B_r_blocks; read_block++){
                for(int read_block_y=0; read_block_y < B_c_blocks; read_block_y++){
                    int tidx_ = read_block * BASE_THREAD_NUM + tidx;
                    int tidx_y_ = read_block_y * BASE_THREAD_NUM + tidx_y;
                    if(tidx_ <B_r && (i * B_r + tidx_ < N) && tidx_y_ < B_c && (j * B_c + tidx_y_ < N)){
                        Sij[tidx_ * B_c + tidx_y_] = exp(Sij[tidx_ * B_c + tidx_y_] - mi[tidx_]);
                    } 
                }
            }
            __syncthreads();
#ifdef TIME
            if(tidx==0 and tidx_y == 0){
                end_time  = clock();
                printf("%f: Pij\n", 1000000.0*(end_time - start_time));
                start_time = clock();
            }
#endif
            for(int read_block=0; read_block < B_r_blocks; read_block++){
                int tidx_ = read_block * BASE_THREAD_NUM + tidx;
                for(int y = 0; y < B_c; y++){
                    if(tidx_ < B_r && (i * B_r + tidx_ < N) && tidx_y == 0 && (j * B_c + y < N)){
                        li[tidx_] += Sij[tidx_ * B_c + y];
                    }  
                }
            }
            __syncthreads();

#ifdef TIME
            if(tidx==0 and tidx_y == 0){
                end_time  = clock();
                printf("%f: lij\n", 1000000.0*(end_time - start_time));
                start_time = clock();
            }
#endif

            for(int read_block=0; read_block < B_r_blocks; read_block++){
                for(int read_block_y=0; read_block_y < d_blocks; read_block_y++){
                    int tidx_ = read_block * BASE_THREAD_NUM + tidx;
                    int tidx_y_ = read_block_y * BASE_THREAD_NUM + tidx_y;
                    for(int y = 0; y < B_c; y++){
                        if(tidx_ < B_r && (i * B_r + tidx_ < N) && tidx_y_ < d){
                            tempPRO[tidx_ * d + tidx_y_] += (Sij[tidx_ * B_c + y] * Vj[y * d + tidx_y_]);
                        }   
                    }
                }
            }
            __syncthreads();
#ifdef TIME
            if(tidx==0 and tidx_y == 0){
                end_time  = clock();
                printf("%f: temp\n", 1000000.0*(end_time - start_time));
                start_time = clock();
            }
#endif
            for(int read_block=0; read_block < B_r_blocks; read_block++){
                int tidx_ = read_block * BASE_THREAD_NUM + tidx;
                if(tidx_ < B_r && i * B_r + tidx_ < N){
                    for(int read_block_y=0; read_block_y < d_blocks; read_block_y++){                   
                        int tidx_y_ = read_block_y * BASE_THREAD_NUM + tidx_y;
                        if(tidx_y_ < d){
                            if(j == 0)
                                Oi[tidx_ * d + tidx_y_] = tempPRO[tidx_ * d + tidx_y_];
                            else
                                Oi[tidx_ * d + tidx_y_] = exp(mi_prev[tidx_] - mi[tidx_]) * Oi[tidx_ * d + tidx_y_] + tempPRO[tidx_ * d + tidx_y_];
                        }
                    }
                }
            }
            __syncthreads();
#ifdef TIME
            if(tidx==0 and tidx_y == 0){
                end_time  = clock();
                printf("%f: Output\n", 1000000.0*(end_time - start_time));
                start_time = clock();
            }
#endif
        }
        __syncthreads();
        for(int read_block=0; read_block < B_r_blocks; read_block++){
            int tidx_ = read_block * BASE_THREAD_NUM + tidx;
            if(tidx_ < B_r){
                if(i * B_r + tidx_ < N && tidx_y == 0){
                    l[i * B_r + tidx_] = mi[tidx_] + log(li[tidx_]);
                }
                for(int read_block_y=0; read_block_y < d_blocks; read_block_y++){                   
                    int tidx_y_ = read_block_y * BASE_THREAD_NUM + tidx_y;
                    if(tidx_y_ < d){ 
                        if(i * B_r + tidx_ < N){
                            out[(i * B_r + tidx_) * d + tidx_y_] = (1.0/li[tidx_]) * Oi[tidx_ * d + tidx_y_];
                        }
                    }
                }
            }
        }
        __syncthreads();
    //}
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

extern "C" {

void launch_flash_attn_fw(
    float* q,
    float* k,
    float* v,
    float* out,
    float* l,
    float* m,
    int batch, int N, int d,
    bool causal_mask,
    hipStream_t stream
) {
    
    // Allocate device memory
    float *d_q, *d_k, *d_v, *d_out, *d_l, *d_m;
    hipMalloc(&d_q, batch * N * d * sizeof(float));
    hipMalloc(&d_k, batch * N * d * sizeof(float));
    hipMalloc(&d_v, batch * N * d * sizeof(float));
    hipMalloc(&d_out, batch * N * d * sizeof(float));    
    hipMalloc(&d_l, batch * N * sizeof(float));
    hipMalloc(&d_m, batch * N * sizeof(float));
    

    // Copy data to the device
    hipMemcpy(d_q, q, batch * N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_k, k, batch * N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, batch * N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, batch * N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_l, l, batch * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m, m, batch * N * sizeof(float), hipMemcpyHostToDevice);

    int B_r = min(MBY4D, d); 
    int T_r = (N + B_r - 1)/ B_r;
    
    dim3 grid_dim(batch, T_r);  // batch_size x num_heads
    dim3 block_dim(BASE_THREAD_NUM, BASE_THREAD_NUM);

    flash_attn_fw<float><<<grid_dim, block_dim, 0, stream>>>(d_q, d_k, d_v, d_out, d_l, d_m, batch, N, d, causal_mask);

    hipMemcpy(out, d_out, batch * N * d * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(l, d_l, batch * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(m, d_m, batch * N * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    
    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        gpuErrchk(err);
      fprintf(stderr, "Flash Attention Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_q);
    hipFree(d_k);
    hipFree(d_v);
    hipFree(d_out);
    hipFree(d_l);
    hipFree(d_m);
    
}

}
}  
} 
